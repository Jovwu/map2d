#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdint> 
#include <vector>
#include <iomanip>
#include <string>
#include <random>
#include <thread>
#include <hip/hip_runtime.h>
#include <thrust/universal_vector.h>
#include <tbb/tbb.h>
#include "camera.h"
#include "mredis.h"

namespace {

using camera::Camera;

std::shared_ptr<RedisSingleton> redis = nullptr;
tbb::task_group tg;

constexpr int PRE_STEP = 3;

using MyCudaFunc = void (camera::Camera::*)(std::uint8_t*, size_t);
MyCudaFunc funcs[] = {&Camera::Up, &Camera::Down, &Camera::Left, &Camera::Right};

std::vector<std::uint8_t> initial_map = {
        1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,2,2,2,2,2,0,0,0,0,3,0,3,0,3,0,0,0,1,
        1,0,0,0,0,0,2,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,2,0,0,0,2,0,0,0,0,3,0,0,0,3,0,0,0,1,
        1,0,0,0,0,0,2,0,0,0,2,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,2,2,0,2,2,0,0,0,0,3,0,3,0,3,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,4,4,4,4,4,4,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,0,4,0,0,0,0,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,0,0,0,0,5,0,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,0,4,0,0,0,0,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,0,4,4,4,4,4,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,4,4,4,4,4,4,4,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,
        1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1
};
thrust::universal_vector<std::uint8_t> map(initial_map.begin(), initial_map.end());

void init(); 
void message_handler(const std::string& message);
__global__ void kernel(Camera camera, std::uint8_t* map, std::uint8_t* canvas);

void init() {
    
    redis = RedisSingleton::GetSingleton();
    redis->Set("test_redis_connect", "success");
    auto val = redis->Get("test_redis_connect");

    if (val)    printf("init redis success");
    else        printf("init redis failed");
    
    tbb::global_control global_limit(tbb::global_control::max_allowed_parallelism, tbb::this_task_arena::max_concurrency());
} 

void message_handler(const std::string& message) {
    
    tg.run([=](){

        Camera msg_camera(message);

        tbb::concurrent_vector<Camera> tasks;
        tasks.reserve(std::size(funcs) * PRE_STEP);

        std::uint8_t* map_ptr = thrust::raw_pointer_cast(initial_map.data());

        tbb::parallel_for(tbb::blocked_range<size_t>(0, std::size(funcs)), [&](const tbb::blocked_range<size_t>& range) {
            for(size_t i = range.begin(); i < range.end(); i++) {
                Camera task_camera = msg_camera;
                for(size_t j = 0; j < PRE_STEP; j++) {
                    (task_camera.*funcs[i])(map_ptr, 24);
                    tasks.emplace_back(task_camera);
                }
            }
        });

        tbb::parallel_for_each(tasks.begin(), tasks.end(), [](Camera& camera) {

            thrust::universal_vector<std::uint8_t> canvas(camera::CANVAS_WIDTH * camera::CANVAS_HEIGHT, 0);
            std::uint8_t* canvas_ptr = thrust::raw_pointer_cast(canvas.data());
            std::uint8_t* map_ptr = thrust::raw_pointer_cast(map.data());
            kernel<<<camera::CANVAS_HEIGHT, camera::CANVAS_WIDTH>>>(camera, map_ptr, canvas_ptr);
            hipDeviceSynchronize();

            std::vector<std::uint8_t> std_canvas(canvas.size());
            thrust::copy(canvas.begin(), canvas.end(), std_canvas.begin());
            redis->Set(camera.ConvertToHashKey(), redis->CovertVectorToValue(std_canvas));
           
        });
    });
  
}

__global__ void kernel(Camera camera, std::uint8_t* map, std::uint8_t* canvas) {

    int x = threadIdx.x;
    int y = blockIdx.x;
 
    int map_x = static_cast<int>(camera.m_position_x), 
        map_y = static_cast<int>(camera.m_position_y),
        // 射线步进长度
        step_x, step_y, 
        side, hit = 0;

    float camera_x = 2 * x / static_cast<float>(camera::CANVAS_WIDTH) - 1,
        // 射线在 X 方向上迈出一步时在 y 方向上行进的单位数
        ray_dir_x = __fmaf_rn(camera.m_plane_x, camera_x, camera.m_dir_x),
        ray_dir_y = __fmaf_rn(camera.m_plane_y, camera_x, camera.m_dir_y),
        // 射线从 1 个 x 边到下一个 x 边或从 1 个 y 边到下一个 y 边必须行进的距离
        delta_dist_x = (ray_dir_x == 0) ? 1e30 : abs(1 / ray_dir_x),
        delta_dist_y = (ray_dir_y == 0) ? 1e30 : abs(1 / ray_dir_y),
        // 射线从其起始位置到第一个 x 边和第一个 y 边必须行进的距离
        side_dist_x, side_dist_y,
        // 射线的长度 
        perp_wall_dist;

    // 根据射线方向决定步进和初始距离
    if (ray_dir_x < 0) {
        step_x      = -1;
        side_dist_x = (camera.m_position_x - map_x) * delta_dist_x;
    } else {
        step_x      = 1;
        side_dist_x = (map_x + 1.0 - camera.m_position_x) * delta_dist_x;
    }
    if (ray_dir_y < 0) {
        step_y      = -1;
        side_dist_y = (camera.m_position_y - map_y) * delta_dist_y;
    } else {
        step_y      = 1;
        side_dist_y = (map_y + 1.0 - camera.m_position_y) * delta_dist_y;
    }

    // DDA: 检测射线撞墙
    while (hit == 0) {
        if (side_dist_x < side_dist_y) {
            side_dist_x += delta_dist_x;
            map_x       += step_x;
            side        = 0;
        } else {
            side_dist_y += delta_dist_y;
            map_y       += step_y;
            side        = 1;
        }
        if (map[static_cast<int>(__fmaf_rn(map_x, map2d::MAP2D_WIDTH, map_y))] > 0) hit = 1;
    }

    if (side == 0) perp_wall_dist = (side_dist_x - delta_dist_x);
    else           perp_wall_dist = (side_dist_y - delta_dist_y);

    int draw_start, draw_end;
    int line_height = static_cast<int>(camera::CANVAS_HEIGHT / perp_wall_dist);
    if (draw_start = (-line_height + camera::CANVAS_HEIGHT) >> 1, draw_start < 0)                  
        draw_start = 0;
    if (draw_end = (line_height + camera::CANVAS_HEIGHT) >> 1, draw_end >= camera::CANVAS_HEIGHT)  
        draw_end = camera::CANVAS_HEIGHT - 1;

    std::uint8_t color = 255;
    switch(map[static_cast<int>(__fmaf_rn(map_x, map2d::MAP2D_WIDTH, map_y))]) {
        case 1:  color = 255;    break; 
        case 2:  color = 128;    break; 
        case 3:  color = 64;     break; 
        case 4:  color = 200;    break; 
        default: color = 100;    break; 
    }

    color >>= side; // side = [0, 1]
    // = if (side == 1) color = color >> 1;

    if (draw_start <= y && y < draw_end) {
        canvas[static_cast<int>(__fmaf_rn(y, camera::CANVAS_WIDTH, x))] = color;
    }
}

} // namespace

int main() {

    init();

    std::thread subscriber_thread([]() {
        while(true) {
            std::this_thread::sleep_for(std::chrono::milliseconds(1));
            redis->Subscribe("server2cuda", message_handler);
        }
    });
    subscriber_thread.join();

    return 0;
}